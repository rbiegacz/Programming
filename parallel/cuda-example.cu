
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square_kernel(float * d_out, float * d_in) {
  int index = threadIdx.x;
  float f = d_in[index];
  d_out[index] = f * f;
}


int main (int argc, char **argv) {
    const int ARRAY_SIZE = 1024;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // filling in initial array of numbers
    float h_in[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; i++) {
    	h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];

    // declaration of GPU mem pointers
    float * d_in;
    float * d_out;


    // allocate GPU memory
    hipMalloc((void**) &d_in, ARRAY_BYTES);
    hipMalloc((void**) &d_out, ARRAY_BYTES);

    // transfer the array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // launch the kernel
    square_kernel<<<1, ARRAY_SIZE>>>(d_out, d_in);

    // copy back the result array to the CPU
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // print out the resulting array
    for (int i =0; i < ARRAY_SIZE; i++) {
    	printf("%f", h_out[i]);
	printf(((i % 4) != 3) ? "\t" : "\n");
    }

    hipFree(d_in);
    hipFree(d_out);
    
    return 0;
}
